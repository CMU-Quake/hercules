#include "hip/hip_runtime.h"
/* -*- C -*- */

/* @copyright_notice_start
 *
 * This file is part of the CMU Hercules ground motion simulator developed
 * by the CMU Quake project.
 *
 * Copyright (C) Carnegie Mellon University. All rights reserved.
 *
 * This program is covered by the terms described in the 'LICENSE.txt' file
 * included with this software package.
 *
 * This program comes WITHOUT ANY WARRANTY; without even the implied warranty
 * of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * 'LICENSE.txt' file for more details.
 *
 *  @copyright_notice_end
 */

#include <inttypes.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "psolve.h"
#include "kernel.h"
#include "quake_util.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


/**
 * For regular simulations use 1e-20.  For performance and scalability,
 * uncomment the immediate return in vector_is_zero() and vector_is_all_zero().
 * Alternatives with each platform underflow precision limit for 'double'
 * may also work, though this have not been thoroughly tested.
 *
 * Known underflow limits:
 * - NICS' Kraken: Limit is ~2.2e-308 --> use 1e-200
 *
 */
#define UNDERFLOW_CAP_STIFFNESS 1e-20


int32_t gpu_get_blocksize(gpu_spec_t *gpuSpecs, char* kernel)
{
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(kernel));

    int computed = gpuSpecs->regs_per_block / attributes.numRegs;
    computed = 1 << (int)floor(log(computed)/log(2));

    return(imin(computed, gpuSpecs->max_threads));
}


__global__  void kernelStiffnessInitLookup(int32_t nharbored,
					   int32_t    myLinearElementsCount,
				int32_t*   myLinearElementsMapperDevice,
				elem_t*    elemTableDevice,
				rev_entry_t* reverseLookupDevice)
{
    int       i;
    int32_t   lnid = (blockIdx.x * blockDim.x) + threadIdx.x; 
    int32_t   lin_eindex;
    elem_t*   elemp;
    int32_t   eindex;

    /* Since number of nodes may not be exactly divisible by block size,
       check that we are not off the end of the node array */
    if (lnid >= nharbored) {
      return;
    }

    rev_entry_t *tableEntry = &(reverseLookupDevice[lnid]);

    memset(tableEntry, 0, sizeof(rev_entry_t));
    
    /* loop on the number of elements */
    for (lin_eindex = 0; lin_eindex < myLinearElementsCount; lin_eindex++) {
      
      eindex = myLinearElementsMapperDevice[lin_eindex];
      elemp  = &(elemTableDevice[eindex]);
      
      for (i = 0; i < 8; i++) {
	if (lnid == elemp->lnid[i]) {
	  tableEntry->lf_indices[tableEntry->count].index = lin_eindex*8 + i;
	  (tableEntry->count)++;
	}
      }
    }
}


/* Stiffness Calc-Force Kernel */
__global__  void kernelStiffnessCalcLocal(int32_t   myLinearElementsCount,
					 int32_t*  myLinearElementsMapperDevice,
					 elem_t*   elemTableDevice,
					 e_t*      eTableDevice,
					 fvector_t* tm1Device,
					 fvector_t* localForceDevice) 
{
    int       i;
    int32_t   eindex;
    int32_t   lin_eindex = (blockIdx.x * blockDim.x) + threadIdx.x; 
    fvector_t curDisp[8];

    register fvector_t localForceReg[8];
    register int32_t   lnidReg[8];
    register e_t       eTableReg;

    /* Since number of elements may not be exactly divisible by block size,
       check that we are not off the end of the element array */
    if (lin_eindex >= myLinearElementsCount) {
      return;
    }

    eindex = myLinearElementsMapperDevice[lin_eindex];
  
    /* Copy node ids and constants from global mem to registers */
    memcpy(lnidReg, elemTableDevice[eindex].lnid, 8*sizeof(int32_t));
    memcpy(&eTableReg, &(eTableDevice[eindex]), sizeof(e_t));
  
    /* Get current displacements */
    for (i = 0; i < 8; i++) {
      memcpy(&(curDisp[i]), tm1Device + lnidReg[i], sizeof(fvector_t));
    }
    
    /* Coefficients for new stiffness matrix calculation */
    if (vector_is_zero( curDisp ) != 0) {
      
      double first_coeff  = -0.5625 * (eTableReg.c2 + 2 * eTableReg.c1);
      double second_coeff = -0.5625 * (eTableReg.c2);
      double third_coeff  = -0.5625 * (eTableReg.c1);
      
      double atu[24];
      double firstVec[24];
      
      aTransposeU( curDisp, atu );
      firstVector( atu, firstVec, first_coeff, second_coeff, third_coeff );
      au( localForceReg, firstVec );
    }

    /* Copy local forces from registers to global mem */
    memcpy(&(localForceDevice[lin_eindex*8]), localForceReg, 
	     8*sizeof(fvector_t));
}


/* Stiffness Add-force Kernel */
__global__  void kernelStiffnessAddLocal(int32_t nharbored,
					 rev_entry_t* reverseLookupDevice,
					 fvector_t* localForceDevice,
					 fvector_t* forceDevice)
{
    int          i;
    int32_t      lnid = (blockIdx.x * blockDim.x) + threadIdx.x; 

    fvector_t*            localForce;
    register rev_entry_t  revReg;
    register fvector_t    nodalForceReg;

    /* Since number of nodes may not be exactly divisible by block size,
       check that we are not off the end of the node array */
    if (lnid >= nharbored) {
      return;
    }

    /* Copy reverse lookup table from global to register */
    memcpy(&revReg, reverseLookupDevice + lnid, sizeof(rev_entry_t));

    /* Copy nodal force from global to register */
    memcpy(&nodalForceReg, forceDevice + lnid, sizeof(fvector_t));

    /* Update forces for this node */
    for (i = 0; i < revReg.count; i++) {
      localForce = &(localForceDevice[revReg.lf_indices[i].index]);

      nodalForceReg.f[0] += localForce->f[0];
      nodalForceReg.f[1] += localForce->f[1];
      nodalForceReg.f[2] += localForce->f[2];
    }

    /* Copy updated nodal force from register to global */
    memcpy(forceDevice + lnid, &nodalForceReg, sizeof(fvector_t));
}


/* -------------------------------------------------------------------------- */
/*                         Efficient Method Utilities                         */
/* -------------------------------------------------------------------------- */


/**
 * For effective stiffness method:
 *
 * Check whether all components of a 3D vector are close to zero,
 * i.e., less than a small threshold.
 *
 * \return 1 when there is at least one "non-zero" component;
 *         0 when all the components are "zero".
 */
__host__ __device__ int vector_is_zero( const fvector_t* v )
{
    /*
     * For scalability studies, uncomment the immediate return.
     */

    /* return 1; */

    int i,j;

    for (i = 0; i < 8; i++) {
        for(j = 0; j < 3; j++){
            if (fabs( v[i].f[j] ) > UNDERFLOW_CAP_STIFFNESS) {
                return 1;
            }
        }
    }

    return 0;
}


__host__ __device__ void aTransposeU( fvector_t* un, double* atu )
{
    double temp[24];
    double u[24];
    int    i, j;

    /* arrange displacement values in an array */
    for (i=0; i<8; i++) {
        for(j=0; j<3; j++) {
            temp[i*3 + j] = un[i].f[j];     /* u1 u2 u3 .... v1 v2 v3 ... z1 z2 z3 */
	}
    }

    reformU( temp, u );

    /* atu[0] = u[0] + u[1] + u[2] + u[3] + u[4] + u[5] + u[6] + u[7]; */
    atu[0]  = 0;
    atu[1]  = -u[0] - u[1] - u[2] - u[3] + u[4] + u[5] + u[6] + u[7];
    atu[2]  = -u[0] - u[1] + u[2] + u[3] - u[4] - u[5] + u[6] + u[7];
    atu[3]  = -u[0] + u[1] - u[2] + u[3] - u[4] + u[5] - u[6] + u[7];
    atu[4]  =  u[0] + u[1] - u[2] - u[3] - u[4] - u[5] + u[6] + u[7];
    atu[5]  =  u[0] - u[1] + u[2] - u[3] - u[4] + u[5] - u[6] + u[7];
    atu[6]  =  u[0] - u[1] - u[2] + u[3] + u[4] - u[5] - u[6] + u[7];
    atu[7]  = -u[0] + u[1] + u[2] - u[3] + u[4] - u[5] - u[6] + u[7];

    /* atu[8] = u[8] + u[9] + u[10] + u[11] + u[12] + u[13] + u[14] + u[15]; */
    atu[8]  = 0;
    atu[9]  = -u[8] - u[9] - u[10] - u[11] + u[12] + u[13] + u[14] + u[15];
    atu[10] = -u[8] - u[9] + u[10] + u[11] - u[12] - u[13] + u[14] + u[15];
    atu[11] = -u[8] + u[9] - u[10] + u[11] - u[12] + u[13] - u[14] + u[15];
    atu[12] =  u[8] + u[9] - u[10] - u[11] - u[12] - u[13] + u[14] + u[15];
    atu[13] =  u[8] - u[9] + u[10] - u[11] - u[12] + u[13] - u[14] + u[15];
    atu[14] =  u[8] - u[9] - u[10] + u[11] + u[12] - u[13] - u[14] + u[15];
    atu[15] = -u[8] + u[9] + u[10] - u[11] + u[12] - u[13] - u[14] + u[15];

    /* atu[16] = u[16] + u[17] + u[18] + u[19] + u[20] + u[21] + u[22] + u[23]; */
    atu[16] = 0;
    atu[17] = -u[16] - u[17] - u[18] - u[19] + u[20] + u[21] + u[22] + u[23];
    atu[18] = -u[16] - u[17] + u[18] + u[19] - u[20] - u[21] + u[22] + u[23];
    atu[19] = -u[16] + u[17] - u[18] + u[19] - u[20] + u[21] - u[22] + u[23];
    atu[20] =  u[16] + u[17] - u[18] - u[19] - u[20] - u[21] + u[22] + u[23];
    atu[21] =  u[16] - u[17] + u[18] - u[19] - u[20] + u[21] - u[22] + u[23];
    atu[22] =  u[16] - u[17] - u[18] + u[19] + u[20] - u[21] - u[22] + u[23];
    atu[23] = -u[16] + u[17] + u[18] - u[19] + u[20] - u[21] - u[22] + u[23];
}

__host__ __device__ void firstVector( const double* atu, 
				      double* finalVector, 
				      double a, 
				      double c, 
				      double b )
{
    finalVector[0] = 0;
    finalVector[1] = b * (atu[19] + atu[1]);
    finalVector[2] = b * (atu[11] + atu[2]);
    finalVector[3] = a * atu[3] + c * (atu[10] + atu[17]);
    finalVector[4] = b * (atu[13] + atu[22] + 2. * atu[4]) / 3.;
    finalVector[5] = ( (a + b) * atu[5] + c * atu[12] ) /3.;
    finalVector[6] = ( (a + b) * atu[6] + c * atu[20] ) /3.;
    finalVector[7] = ( (a + 2.*b) * atu[7] ) / 9.;

    finalVector[8] = 0;
    finalVector[9] = b * (atu[18] + atu[9]);
    finalVector[10] = a * atu[10] + c * (atu[3] + atu[17]);
    finalVector[11] = b * (atu[11] + atu[2]);
    finalVector[12] = ( (a + b) * atu[12] + c * atu[5] ) / 3.;
    finalVector[13] = b * (atu[4] + atu[22] + 2. * atu[13]) / 3.;
    finalVector[14] = ( (a + b) * atu[14] + c * atu[21] ) /3.;
    finalVector[15] = (a + 2. * b) * atu[15] / 9.;

    finalVector[16] = 0;
    finalVector[17] = a * atu[17] + c * (atu[3] + atu[10]);
    finalVector[18] = b * (atu[18] + atu[9]);
    finalVector[19] = b * (atu[19] + atu[1]);
    finalVector[20] = ( (a + b) * atu[20] + c * atu[6] ) / 3.;
    finalVector[21] = ( (a + b) * atu[21] + c * atu[14] ) / 3.;
    finalVector[22] = b * ( atu[4] + atu[13] + 2. * atu[22]) / 3.;
    finalVector[23] = (a + 2. * b) * atu[23] / 9.;
}


__host__ __device__ void au( fvector_t* resVec, const double* u )
{
    int    i, j;
    double finVec[24];
    double temp[24];


    finVec[0]  = u[0]  - u[1] - u[2] - u[3] + u[4] + u[5] + u[6] - u[7];
    finVec[1]  = u[0]  - u[1] - u[2] + u[3] + u[4] - u[5] - u[6] + u[7];
    finVec[2]  = u[0]  - u[1] + u[2] - u[3] - u[4] + u[5] - u[6] + u[7];
    finVec[3]  = u[0]  - u[1] + u[2] + u[3] - u[4] - u[5] + u[6] - u[7];
    finVec[4]  = u[0]  + u[1] - u[2] - u[3] - u[4] - u[5] + u[6] + u[7];
    finVec[5]  = u[0]  + u[1] - u[2] + u[3] - u[4] + u[5] - u[6] - u[7];
    finVec[6]  = u[0]  + u[1] + u[2] - u[3] + u[4] - u[5] - u[6] - u[7];
    finVec[7]  = u[0]  + u[1] + u[2] + u[3] + u[4] + u[5] + u[6] + u[7];

    finVec[8]  = u[8]  - u[9] - u[10] - u[11] + u[12] + u[13] + u[14] - u[15];
    finVec[9]  = u[8]  - u[9] - u[10] + u[11] + u[12] - u[13] - u[14] + u[15];
    finVec[10] = u[8]  - u[9] + u[10] - u[11] - u[12] + u[13] - u[14] + u[15];
    finVec[11] = u[8]  - u[9] + u[10] + u[11] - u[12] - u[13] + u[14] - u[15];
    finVec[12] = u[8]  + u[9] - u[10] - u[11] - u[12] - u[13] + u[14] + u[15];
    finVec[13] = u[8]  + u[9] - u[10] + u[11] - u[12] + u[13] - u[14] - u[15];
    finVec[14] = u[8]  + u[9] + u[10] - u[11] + u[12] - u[13] - u[14] - u[15];
    finVec[15] = u[8]  + u[9] + u[10] + u[11] + u[12] + u[13] + u[14] + u[15];

    finVec[16] = u[16] - u[17] - u[18] - u[19] + u[20] + u[21] + u[22] - u[23];
    finVec[17] = u[16] - u[17] - u[18] + u[19] + u[20] - u[21] - u[22] + u[23];
    finVec[18] = u[16] - u[17] + u[18] - u[19] - u[20] + u[21] - u[22] + u[23];
    finVec[19] = u[16] - u[17] + u[18] + u[19] - u[20] - u[21] + u[22] - u[23];
    finVec[20] = u[16] + u[17] - u[18] - u[19] - u[20] - u[21] + u[22] + u[23];
    finVec[21] = u[16] + u[17] - u[18] + u[19] - u[20] + u[21] - u[22] - u[23];
    finVec[22] = u[16] + u[17] + u[18] - u[19] + u[20] - u[21] - u[22] - u[23];
    finVec[23] = u[16] + u[17] + u[18] + u[19] + u[20] + u[21] + u[22] + u[23];

    reformF( finVec, temp );

    for (j = 0; j<8; j++)
    {
        for (i = 0; i<3; i++)
        {
            resVec[j].f[i] += temp[j*3 + i];
        }
    }
}


__host__ __device__ void reformF( const double* u, double* newU )
{
    newU[0]  = u[0];
    newU[1]  = u[8];
    newU[2]  = u[16];
    newU[3]  = u[1];
    newU[4]  = u[9];
    newU[5]  = u[17];
    newU[6]  = u[2];
    newU[7]  = u[10];
    newU[8]  = u[18];
    newU[9]  = u[3];
    newU[10] = u[11];
    newU[11] = u[19];
    newU[12] = u[4];
    newU[13] = u[12];
    newU[14] = u[20];
    newU[15] = u[5];
    newU[16] = u[13];
    newU[17] = u[21];
    newU[18] = u[6];
    newU[19] = u[14];
    newU[20] = u[22];
    newU[21] = u[7];
    newU[22] = u[15];
    newU[23] = u[23];
}

__host__ __device__ void reformU( const double* u, double* newU )
{
    newU[0]  = u[0];
    newU[1]  = u[3];
    newU[2]  = u[6];
    newU[3]  = u[9];
    newU[4]  = u[12];
    newU[5]  = u[15];
    newU[6]  = u[18];
    newU[7]  = u[21];
    newU[8]  = u[1];
    newU[9]  = u[4];
    newU[10] = u[7];
    newU[11] = u[10];
    newU[12] = u[13];
    newU[13] = u[16];
    newU[14] = u[19];
    newU[15] = u[22];
    newU[16] = u[2];
    newU[17] = u[5];
    newU[18] = u[8];
    newU[19] = u[11];
    newU[20] = u[14];
    newU[21] = u[17];
    newU[22] = u[20];
    newU[23] = u[23];
}
