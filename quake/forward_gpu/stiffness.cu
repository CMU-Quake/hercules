/* -*- C -*- */

/* @copyright_notice_start
 *
 * This file is part of the CMU Hercules ground motion simulator developed
 * by the CMU Quake project.
 *
 * Copyright (C) Carnegie Mellon University. All rights reserved.
 *
 * This program is covered by the terms described in the 'LICENSE.txt' file
 * included with this software package.
 *
 * This program comes WITHOUT ANY WARRANTY; without even the implied warranty
 * of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * 'LICENSE.txt' file for more details.
 *
 *  @copyright_notice_end
 */

#include <inttypes.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "psolve.h"
#include "nonlinear.h" //NEEDS TO BE HERE FOR NONLINEAR TO RUN
#include "stiffness.h"
#include "quake_util.h"
#include "kernel.h"
#include "util.h"
#include "timers.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


/* -------------------------------------------------------------------------- */
/*                             Global Variables                               */
/* -------------------------------------------------------------------------- */

static int32_t    myLinearElementsCount;
static int32_t   *myLinearElementsMapper;
static int32_t   *myLinearElementsMapperDevice;
static rev_entry_t   *reverseLookupLinearDevice;

static int initLookupBlockSize;
static int calcForceBlockSize;
static int addForceBlockSize;


/* -------------------------------------------------------------------------- */
/*          Initialization of parameters for nonlinear compatibility          */
/* -------------------------------------------------------------------------- */

/**
 * Counts the number of nonlinear elements in my local mesh
 */
void linear_elements_count(int32_t myID, mesh_t *myMesh) {

    int32_t eindex;
    int32_t count = 0;

    for (eindex = 0; eindex < myMesh->lenum; eindex++) {

        if ( isThisElementNonLinear(myMesh, eindex) == NO ) {
            count++;
        }
    }

    if ( count > myMesh-> lenum ) {
        fprintf(stderr,"Thread %d: linear_elements_count: "
                "more elements than expected\n", myID);
        MPI_Abort(MPI_COMM_WORLD, ERROR);
        exit(1);
    }

    myLinearElementsCount = count;

    return;
}


/**
 * Re-counts and stores the nonlinear element indices to a static local array
 * that will serve as mapping tool to the local mesh elements table.
 */
void linear_elements_mapping(int32_t myID, mesh_t *myMesh) {

    int32_t eindex;
    int32_t count = 0;

    XMALLOC_VAR_N(myLinearElementsMapper, int32_t, myLinearElementsCount);

    for (eindex = 0; eindex < myMesh->lenum; eindex++) {

        if ( isThisElementNonLinear(myMesh, eindex) == NO ) {
            myLinearElementsMapper[count] = eindex;
            count++;
        }
    }

    if ( count != myLinearElementsCount ) {
        fprintf(stderr,"Thread %d: linear_elements_mapping: "
                "more elements than the count\n", myID);
        MPI_Abort(MPI_COMM_WORLD, ERROR);
        exit(1);
    }

    return;
}


void stiffness_init(int32_t myID, mesh_t *myMesh, mysolver_t* mySolver)
{
    linear_elements_count(myID, myMesh);
    linear_elements_mapping(myID, myMesh);
    
    /* Allocate device memory */
    if (hipMalloc((void**)&myLinearElementsMapperDevice, 
		   myLinearElementsCount * sizeof(int32_t)) != hipSuccess) {
        fprintf(stderr, "Thread %d: Failed to allocate mapper memory\n", myID);
        MPI_Abort(MPI_COMM_WORLD, ERROR);
        exit(1);
    }
    if (hipMalloc((void**)&(reverseLookupLinearDevice), 
    		   myMesh->nharbored * sizeof(rev_entry_t)) != hipSuccess) {
            fprintf(stderr, "Thread %d: Failed to allocate reverseLookup memory\n", 
    		myID);
            MPI_Abort(MPI_COMM_WORLD, ERROR);
            exit(1);
    }

    /* Copy linear element mapper to device */
    if (hipMemcpy(myLinearElementsMapperDevice, myLinearElementsMapper, 
		   myLinearElementsCount * sizeof(int32_t),  
		   hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Thread %d: Failed to copy mapper to device - %s\n", 
		myID, hipGetErrorString(hipGetLastError()));
        MPI_Abort(MPI_COMM_WORLD, ERROR);
        exit(1);
    }

    /* Dynamically calculate optimum block size for each kernel */
    initLookupBlockSize = gpu_get_blocksize(mySolver->gpu_spec,
					    (char *)kernelInitLinearLookup);
    calcForceBlockSize = gpu_get_blocksize(mySolver->gpu_spec,
					   (char *)kernelStiffnessCalcLocal);
    addForceBlockSize = gpu_get_blocksize(mySolver->gpu_spec,
					  (char *)kernelAddLocalForces);

    if (myID == 0) {
      fprintf(stderr, "!!!!! computed block sizes = %d, %d, %d\n", 
	      initLookupBlockSize, calcForceBlockSize, addForceBlockSize);
    }

    /* Create reverse lookup table */
    int blocksize = initLookupBlockSize;
    int gridsize = (myMesh->nharbored / blocksize) + 1;
    hipGetLastError();
    kernelInitLinearLookup<<<gridsize, blocksize>>>(mySolver->gpuDataDevice,
						    myLinearElementsCount,
						    myLinearElementsMapperDevice,
						    reverseLookupLinearDevice);

    hipDeviceSynchronize();

    hipError_t cerror = hipGetLastError();
    if (cerror != hipSuccess) {
      fprintf(stderr, "Thread %d: Init linear lookup kernel - %s\n", myID, 
	      hipGetErrorString(cerror));
      MPI_Abort(MPI_COMM_WORLD, ERROR);
      exit(1);
    }

    return;
}


void stiffness_delete(int32_t myID) {
    /* Free stiffness module memory */
    free(myLinearElementsMapper);

    /* Free device memory */
    hipFree(myLinearElementsMapperDevice);
    hipFree(reverseLookupLinearDevice);

    return;
}


/* -------------------------------------------------------------------------- */
/*                       Stiffness Contribution Methods                       */
/* -------------------------------------------------------------------------- */


/**
 * Compute and add the force due to the element stiffness matrices.
 *
 * \param myMesh   Pointer to the solver mesh structure.
 * \param mySolver Pointer to the solver main data structures.
 * \param theK1    First stiffness matrix (K1).
 * \param theK2    Second stiffness matrix (K2).
 */
void compute_addforce_conventional( mesh_t* myMesh, mysolver_t* mySolver, 
				    fmatrix_t (*theK1)[8], fmatrix_t (*theK2)[8] )
{
    fvector_t localForce[8];
    int       i, j;
    int32_t   eindex;
    int32_t   lin_eindex;

    /* loop on the number of elements */
    for (lin_eindex = 0; lin_eindex < myLinearElementsCount; lin_eindex++) {

        elem_t* elemp;
        e_t*    ep;

        eindex = myLinearElementsMapper[lin_eindex];
        elemp  = &myMesh->elemTable[eindex];
        ep     = &mySolver->eTable[eindex];

        /* step 1: calculate the force due to the element stiffness */
        memset( localForce, 0, 8 * sizeof(fvector_t) );

        /* contribution by node j to node i */
        for (i = 0; i < 8; i++)
        {
            fvector_t* toForce = &localForce[i];

            for (j = 0; j < 8; j++)
            {
                int32_t    nodeJ  = elemp->lnid[j];
                fvector_t* myDisp = mySolver->tm1 + nodeJ;

                /*
		 * contributions by the stiffnes/damping matrix
		 * contribution by ( - deltaT_square * Ke * Ut )
		 * But if myDisp is zero avoids multiplications
		 */
                if ( vector_is_all_zero( myDisp ) != 0 ) {
                    MultAddMatVec( &theK1[i][j], myDisp, -ep->c1, toForce );
                    MultAddMatVec( &theK2[i][j], myDisp, -ep->c2, toForce );
                }
            }
        }

        /* step 2: sum up my contribution to my vertex nodes */
        for (i = 0; i < 8; i++) {
            int32_t    lnid       = elemp->lnid[i];
            fvector_t* nodalForce = mySolver->force + lnid;

            nodalForce->f[0] += localForce[i].f[0];
            nodalForce->f[1] += localForce[i].f[1];
            nodalForce->f[2] += localForce[i].f[2];
        }
    } /* for all the elements */
}


/**
 * Compute and add the force due to the element stiffness matrices with the effective method.
 */
void compute_addforce_effective_cpu( mesh_t* myMesh, mysolver_t* mySolver )
{
    /* \TODO use mu_and_lamda to compute first,second and third coefficients */

    fvector_t localForce[8];
    fvector_t curDisp[8];
    int       i;
    int32_t   eindex;
    int32_t   lin_eindex;

    /* loop on the number of elements */
    for (lin_eindex = 0; lin_eindex < myLinearElementsCount; lin_eindex++) {

        elem_t* elemp;
        e_t*    ep;

        eindex = myLinearElementsMapper[lin_eindex];
        elemp  = &myMesh->elemTable[eindex];
        ep     = &mySolver->eTable[eindex];

        memset( localForce, 0, 8 * sizeof(fvector_t) );

        for (i = 0; i < 8; i++) {
            int32_t    lnid = elemp->lnid[i];
            fvector_t* tm1Disp = mySolver->tm1 + lnid;
//	    fvector_t* tm2Disp = mySolver->tm2 + lnid;

            curDisp[i].f[0] = tm1Disp->f[0];
            curDisp[i].f[1] = tm1Disp->f[1];
            curDisp[i].f[2] = tm1Disp->f[2];

        }

        /* Coefficients for new stiffness matrix calculation */
        if (vector_is_zero( curDisp ) != 0) {

            double first_coeff  = -0.5625 * (ep->c2 + 2 * ep->c1);
            double second_coeff = -0.5625 * (ep->c2);
            double third_coeff  = -0.5625 * (ep->c1);

            double atu[24];
            double firstVec[24];

            aTransposeU( curDisp, atu );
            firstVector( atu, firstVec, first_coeff, second_coeff, third_coeff );
            au( localForce, firstVec );
        }

        for (i = 0; i < 8; i++) {
            int32_t lnid          = elemp->lnid[i];;
            fvector_t* nodalForce = mySolver->force + lnid;

            nodalForce->f[0] += localForce[i].f[0];
            nodalForce->f[1] += localForce[i].f[1];
            nodalForce->f[2] += localForce[i].f[2];
        }
    } /* for all the elements */
}


/**
 * Compute and add the force due to the element stiffness matrices with 
   the effective method.
 */
void compute_addforce_effective_gpu( int32_t myID, 
				     mesh_t* myMesh, 
				     mysolver_t* mySolver )
{
    /* Copy working data to device */
    hipMemcpy(mySolver->gpuData->forceDevice, mySolver->force, 
	       myMesh->nharbored * sizeof(fvector_t), hipMemcpyHostToDevice);

    int blocksize = calcForceBlockSize;
    int gridsize = (myLinearElementsCount / blocksize) + 1;
    hipGetLastError();
    kernelStiffnessCalcLocal<<<gridsize, blocksize>>>(mySolver->gpuDataDevice,
						      myLinearElementsCount, 
						      myLinearElementsMapperDevice);

    hipDeviceSynchronize();

    hipError_t cerror = hipGetLastError();
    if (cerror != hipSuccess) {
      fprintf(stderr, "Thread %d: Calc stiffness local kernel - %s\n", myID, 
	      hipGetErrorString(cerror));
      MPI_Abort(MPI_COMM_WORLD, ERROR);
      exit(1);
    }

    blocksize = addForceBlockSize;
    gridsize = ((myMesh->nharbored) / blocksize) + 1;
    hipGetLastError();
    kernelAddLocalForces<<<gridsize, blocksize>>>(mySolver->gpuDataDevice);
    hipDeviceSynchronize();

    cerror = hipGetLastError();
    if (cerror != hipSuccess) {
      fprintf(stderr, "Thread %d: Add stiffness local kernel - %s\n", myID, 
	      hipGetErrorString(cerror));
      MPI_Abort(MPI_COMM_WORLD, ERROR);
      exit(1);
    }

    /* Copy working data back to host */
    hipMemcpy(mySolver->force, mySolver->gpuData->forceDevice,
    	       myMesh->nharbored * sizeof(fvector_t), hipMemcpyDeviceToHost);

    return;
}

